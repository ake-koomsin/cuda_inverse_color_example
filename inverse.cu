#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "utils.h"

__global__ void inverseKernel(const unsigned char *src, unsigned char *dst, size_t width, size_t height, size_t numberOfComponents) {
  int col = blockDim.x * blockIdx.x + threadIdx.x;
  int row = blockDim.y * blockIdx.y + threadIdx.y;

  int index = width * numberOfComponents * row + col;

  // int index = blockDim.x * blockIdx.x + threadIdx.x;

  if (index < width * height * numberOfComponents) {
    dst[index] = 255 - src[index];
    // __syncthreads();
  }

}

extern "C" void inverseImage(const RawImage *src, RawImage *dst) {
  if (src == NULL || src->pixels == NULL || dst == NULL ) {
    return;
  }

  size_t size = src->width * src->height * src->numberOfComponents;

  dst->width = src->width;
  dst->height = src->height;
  dst->numberOfComponents = src->numberOfComponents;
  free(dst->pixels);
  dst->pixels = (unsigned char *)malloc(size);

  unsigned char *srcPixel = src->pixels;
  unsigned char *dstPixel = dst->pixels;

  unsigned char *deviceSrcPixel;
  unsigned char *deviceDstPixel;
  
  hipMalloc((void**)&deviceSrcPixel, size);
  hipMalloc((void**)&deviceDstPixel, size);

  hipMemcpy(deviceSrcPixel, srcPixel, size, hipMemcpyHostToDevice);

  // int threadsPerBlock = 256;
  // int blocksPerGrid = (size) / threadsPerBlock;
  dim3 dimBlock(16, 16);
  dim3 dimGrid((src->width * src->numberOfComponents + dimBlock.x - 1)  / dimBlock.x, (src->height + dimBlock.y - 1) / dimBlock.y);
  inverseKernel<<<dimGrid, dimBlock>>>(deviceSrcPixel, deviceDstPixel, src->width, src->height, src->numberOfComponents);

  hipMemcpy(dstPixel, deviceDstPixel, size, hipMemcpyDeviceToHost);

  hipFree(deviceSrcPixel);
  hipFree(deviceDstPixel);

}



